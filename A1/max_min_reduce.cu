
#include <hip/hip_runtime.h>
#include<iostream>
#define N 20000
#define BLOCK_SIZE 1024
using namespace std;

__global__ void ReduceMax(int n, int *input, int *d_max){
__shared__ int sh[BLOCK_SIZE];
int tid = threadIdx.x;
int myid = threadIdx.x + blockDim.x * blockIdx.x;
if(tid<n)
sh[tid] = input[myid];
else
sh[tid] = 0;
__syncthreads();
for(int i = blockDim.x/2 ; i>0;i>>=1){
if(tid<i)
sh[tid] = max(sh[tid], sh[tid +i]);  //replace by min for minimum and set sh[tid] = N+1
__syncthreads();
}
if(tid==0)
atomicMax(d_max, sh[0]);
}


int main(){
int num_blocks;
int *h, *h_max;
h=(int *)malloc(sizeof(int)*N);
h_max=(int *)malloc(sizeof(int));
for(int i =0;i<N;i++){
h[i] = i+1;
}
int *d_h, *d_max;
if(N/BLOCK_SIZE ==0) num_blocks = 1;
else if(sqrt(N) !=0)
num_blocks = N/BLOCK_SIZE + 1;
else
num_blocks = N/BLOCK_SIZE;
hipMalloc((void **)&d_h, sizeof(int)*N);
hipMalloc((void **)&d_max, sizeof(int));
hipMemcpy(d_h, h, sizeof(int)*N, hipMemcpyHostToDevice);
ReduceMax<<<num_blocks, BLOCK_SIZE>>>(N, d_h, d_max);
hipMemcpy(h_max, d_max, sizeof(int), hipMemcpyDeviceToHost);
cout<<h_max[0]<<endl;
}
