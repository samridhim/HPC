
#include <hip/hip_runtime.h>
#include<stdio.h>
#define N 2000000
#define BLOCK_SIZE 1024
//using namespace std;

__global__ void ReduceMin(int n, int *input, int *output){
__shared__ int sh[BLOCK_SIZE];
int tid = threadIdx.x;
int myId = threadIdx.x + blockIdx.x*blockDim.x;
if(tid<BLOCK_SIZE)
sh[tid] = input[myId];
else
sh[tid] = INT_MAX;
__syncthreads();
for(int i = blockDim.x/2; i>0;i>>=1){
if(tid<i)
{
if(sh[tid]>sh[tid+i])   // sh[tid]<sh[tid +i] for max
sh[tid] = atomicMin(&sh[tid+i], sh[tid]);  //atomicMax for max
else
sh[tid] = sh[tid];
}
__syncthreads();
}
if(tid==0)
output[blockIdx.x] = sh[0];
}

int main(){
int num_blocks;
if(N%BLOCK_SIZE!=0)
 num_blocks = N/BLOCK_SIZE+1;
else if(N/BLOCK_SIZE==0)
num_blocks =1;
else
num_blocks= N/BLOCK_SIZE;
int *h = (int*)malloc(sizeof(int)*N);
int *d_h, *d_temp;
int *h_temp = (int *) malloc(sizeof(int)*1);
hipMalloc((void **)&d_h, sizeof(int)*N);
hipMalloc((void **)&d_temp, sizeof(int)*num_blocks);
for(int i =0;i<N;i++)
h[i] = i+1;
hipMemcpy(d_h, h, sizeof(int)*N, hipMemcpyHostToDevice);
ReduceMin<<<num_blocks,BLOCK_SIZE>>>(BLOCK_SIZE, d_h, d_temp);
hipMemcpy(h, d_temp, sizeof(int)*num_blocks, hipMemcpyDeviceToHost);
int maxx = INT_MAX;  //INT_MIN for max
for(int i =0;i<num_blocks;i++){
if(h[i]<maxx &&h[i]!=0)  //h[i]>maxx for max
maxx = h[i];
}
printf("%d", maxx);
hipFree(d_h);
hipFree(d_temp);
}
