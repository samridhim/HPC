
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10
#define BLOCK_SIZE 512
//Note: N should always be in powers of 2 (like 2, 4, 8, 16, 32, ...) -Mohit Agrawal
int main()
{
 //int num_blocks = N/BLOCK_SIZE;
 int *h_a = (int *) malloc(sizeof(int) * N);
 for(int i =0;i<N;i++){
 h_a[i] = 5;
 printf("%d", h_a[i]);
 }
 int *d_a;
 hipMalloc((void **)&d_a, sizeof(int) *N);
 hipMemcpy(&d_a, h_a, sizeof(int)*N, hipMemcpyHostToDevice);
  h_a = (int *) malloc(sizeof(int));
  hipMemcpy(h_a, d_a, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d", h_a);
}
