
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#include "omp.h"


__global__ void square_reduce(float *n){
n[threadIdx.x] = n[threadIdx.x] * n[threadIdx.x];
}
__global__ void reduce_diff(float *n, float mean){
n[threadIdx.x] = n[threadIdx.x] - mean;
}
__global__ void reduce_max(float *n){
int tid = threadIdx.x;

int stepsize = 1;
int num_threads = gridDim.x;

while(num_threads>0){
	if(tid<num_threads){
		int fst = tid*stepsize*2;
		int snd = fst + stepsize;
		n[fst] += n[snd];
	}
	stepsize<<=1;
	num_threads>>=1;
}
}

int main()
{
float start = omp_get_wtime();
int N = 10;
srand(time(0));
int actual_n = N;
while((N&(N-1))!=0){
N = N+1;
}
printf("%d, %d", actual_n, N);
printf("\n");
float *h_arr, *h_max;
h_arr = (float*)malloc(sizeof(float)*N);
h_max = (float*)malloc(sizeof(float)*N);
for(int i =0;i<N;i++){
if(i<actual_n)
h_arr[i] = 5;
else
h_arr[i] = 0;
}
for(int i =0;i<actual_n;i++){
printf("%0.02f ", h_arr[i]);
}
printf("\n");
float *d_arr;
hipMalloc((void **)&d_arr, sizeof(float)*N);

hipMemcpy(d_arr, h_arr, sizeof(float)*N, hipMemcpyHostToDevice);

reduce_max<<<2, N/2>>>(d_arr);

hipDeviceSynchronize();
hipMemcpy(h_max, d_arr, sizeof(float)*N, hipMemcpyDeviceToHost);
printf("Sum is %0.02f\n", h_max[0]);
printf("Mean is %f\n", (float)h_max[0]/actual_n);
float *h_std_dev_arr;
float *d_std_dev_arr;
h_std_dev_arr = (float *)malloc(sizeof(float)*N);
hipMalloc((void **)&d_std_dev_arr, sizeof(float)*N);
for(int i =0;i<N;i++){
if(i<actual_n)
h_std_dev_arr[i] =h_arr[i];
else
h_std_dev_arr[i] =(float)h_max[0]/actual_n;
}
for(int i =0;i<N;i++){
printf("%0.02f ",h_std_dev_arr[i]);
}

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
float mean = (float)h_max[0]/actual_n;
reduce_diff<<<1, N>>>(d_std_dev_arr,mean);
hipMemcpy(h_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

printf("\n");
for(int i =0;i<N;i++)
printf("%0.02f ",h_std_dev_arr[i]);

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
square_reduce<<<1,N>>>(d_std_dev_arr);
hipMemcpy(h_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

printf("\n");
for(int i =0;i<N;i++)
printf("%0.02f ",h_std_dev_arr[i]);

float *h_sum_std_dev_arr;
h_sum_std_dev_arr = (float*)malloc(sizeof(float)*N);

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
reduce_max<<<1, N/2>>>(d_std_dev_arr);
hipMemcpy(h_sum_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

printf("\n");
printf("%0.02f\n",h_sum_std_dev_arr[0]);

printf("Variance is %0.02f\n",h_sum_std_dev_arr[0]/actual_n);
printf("Std Deviance is %0.02f\n", sqrt(h_sum_std_dev_arr[0]/actual_n));
printf("Time taken %f\n", omp_get_wtime() - start);
return 0;  
}


