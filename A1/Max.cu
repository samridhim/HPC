#include "hip/hip_runtime.h"
#include <stdio.h>
#include<omp.h>
#define BLOCK_SIZE 1024
__global__ void FindSquare(float *input){
int tid = threadIdx.x + blockIdx.x *blockDim.x;
input[tid] *= input[tid]; 
}
__global__ void FindDiff(float *input, float mean, float n){
int tid = threadIdx.x + blockIdx.x *blockDim.x;
if(tid<n)
input[tid] -= mean; 
else
input[tid] =0;
}
__global__ void FindSum(float* input, float *output,int n)
{
	__shared__ int sh[BLOCK_SIZE];		
	int myId = threadIdx.x + blockDim.x * blockIdx.x;
  	int tid = threadIdx.x;
	if(tid<BLOCK_SIZE)
	sh[tid] = input[myId];
	else
	sh[tid] = 0;
	__syncthreads();
	for(unsigned int s = blockDim.x/2 ; s>0 ; s >>=1){

	if(tid<s){
	sh[tid] += sh[tid +s];
	}
	__syncthreads();	
	}
        if(tid==0)
        output[blockIdx.x] = sh[0];
}
int main(int argc, char *argv[])
{
	float start = omp_get_wtime();
	int N = 20000; //2,00,000 elements to be summed
	
 	int num_blocks;
	time_t t;
	srand((unsigned) time(&t));
	int actual_n = N;
	float *h;
	h = (float*)malloc(N*sizeof(float));
	for(int i=0; i<actual_n; i++)
	{
		h[i] = 10;
	}
	printf("\n");
        printf("Elements #: %d\n", actual_n);
	float* d, *d_temp;
        float *d_final;
	hipMalloc(&d, N*sizeof(float));
        if(N/BLOCK_SIZE ==0) 
	{
	num_blocks = 1;
	}
	else if(N%BLOCK_SIZE!=0)
	num_blocks= N/BLOCK_SIZE +1;	
	else
	num_blocks = N/BLOCK_SIZE;
	hipMemcpy(d, h, N*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&d_temp, num_blocks*sizeof(float));
        hipMalloc(&d_final, num_blocks*sizeof(float));
	FindSum<<<num_blocks, BLOCK_SIZE>>>(d, d_temp,num_blocks);
	FindSum <<<num_blocks, BLOCK_SIZE>>>(d_temp,d_final,num_blocks);
	float *result;
	result = (float*)malloc(sizeof(float));
	hipMemcpy(result, d_final, sizeof(float), hipMemcpyDeviceToHost);
	printf("Sum is: %0.02f \n", result[0]);
	printf("Mean is %0.02f \n", (double)result[0]/actual_n);	
	hipMemcpy(d, h, N*sizeof(float), hipMemcpyHostToDevice);
	FindDiff<<<num_blocks, BLOCK_SIZE>>>(d, result[0]/actual_n,actual_n);
	hipMemcpy(h, d, N*sizeof(float), hipMemcpyDeviceToHost);
	hipMemcpy(d,h, N*sizeof(float), hipMemcpyHostToDevice);
	FindSquare<<<num_blocks, BLOCK_SIZE>>>(d);
	hipMemcpy(h,d, N*sizeof(float), hipMemcpyDeviceToHost);	
	hipMemcpy(d, h, N*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc(&d_temp, num_blocks*sizeof(float));
        hipMalloc(&d_final, num_blocks*sizeof(float));
	FindSum<<<num_blocks, BLOCK_SIZE>>>(d, d_temp,actual_n);
	FindSum <<<num_blocks, BLOCK_SIZE>>>(d_temp,d_final,num_blocks);
	hipMemcpy(result, d_final, sizeof(float), hipMemcpyDeviceToHost);
	printf("Variance is: %0.02f \n", (double)result[0]/actual_n);
	printf("Standard Deviation is: %0.02f \n", sqrt((double)result[0]/actual_n));
	printf("Time taken : %f\n", omp_get_wtime()-start);	
	hipFree(d);
	free(h);
	return 0;
}
