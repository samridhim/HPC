#include "hip/hip_runtime.h"
#include<stdio.h>
#include<time.h>
#include "omp.h"


__global__ void square_reduce(float *n){
int myId = threadIdx.x + blockDim.x * blockIdx.x;
int tid = threadIdx.x;
	for(unsigned int s = blockDim.x/2 ; s>0 ; s >>=1){

	if(tid<s){
	n[myId] *= n[myId];
	}
	__syncthreads();	
	}
	if(tid ==0)
	n[blockIdx.x] = n[myId];
}
__global__ void reduce_diff(float *n, float mean){
int myId = threadIdx.x + blockDim.x * blockIdx.x;
  	int tid = threadIdx.x;
	for(unsigned int s = blockDim.x/2 ; s>0 ; s >>=1){

	if(tid<s){
	n[myId] = n[myId] - mean;
	}
	__syncthreads();	
	}
	if(tid ==0)
	n[blockIdx.x] = n[myId];
}
__global__ void reduce_max(float *n){

  int index = blockDim.x * blockIdx.x + threadIdx.x, output_index = blockIdx.x;
  extern __shared__ float temp_arr[];
  if(index<n)
    temp_arr[threadIdx.x] = arr[index];
  else
    temp_arr[threadIdx.x] = 0;
  __syncthreads();

  for(int i=NUM_THREADS_PER_BLOCK/2; i>0; i/=2)
    {
      if(threadIdx.x<i)
	temp_arr[threadIdx.x] += temp_arr[threadIdx.x+i];
      __syncthreads();
    }

  if(threadIdx.x==0)
    b[output_index] = temp_arr[0];
}

int main()
{
float start = omp_get_wtime();
int N = 1024;

srand(time(0));
int actual_n = N;
while((N&(N-1))!=0){
N = N+1;
}
//printf("%d, %d", actual_n, N);
//printf("\n");
float *h_arr, *h_max;
h_arr = (float*)malloc(sizeof(float)*N);
h_max = (float*)malloc(sizeof(float)*N);
for(int i =0;i<N;i++){
if(i<actual_n)
h_arr[i] = 5;
else
h_arr[i] = 0;
}
//for(int i =0;i<actual_n;i++){
//printf("%0.02f ", h_arr[i]);
//}
printf("\n");
float *d_arr,*d_intermediate;
hipMalloc((void **)&d_arr, sizeof(float)*N);
hipMalloc((void **)&d_intermediate, sizeof(float)*2);
hipMemcpy(d_intermediate, h_arr, sizeof(float)*N, hipMemcpyHostToDevice);
reduce_max<<<dim3(2,1,1), N>>>(d_intermediate);
hipMemcpy(d_arr, d_intermediate, sizeof(float)*N, hipMemcpyHostToDevice);
reduce_max<<<dim3(1,1,1), N>>>(d_arr);
hipDeviceSynchronize();
hipMemcpy(h_max, d_arr, sizeof(float)*N, hipMemcpyDeviceToHost);
printf("Sum is %0.02f\n", h_max[0]);
printf("Mean is %f\n", (float)h_max[0]/actual_n);
/*float *h_std_dev_arr;
float *d_std_dev_arr;
h_std_dev_arr = (float *)malloc(sizeof(float)*N);
hipMalloc((void **)&d_std_dev_arr, sizeof(float)*N);
for(int i =0;i<N;i++){
if(i<actual_n)
h_std_dev_arr[i] =h_arr[i];
else
h_std_dev_arr[i] =(float)h_max[0]/actual_n;
}
//for(int i =0;i<N;i++){
//printf("%0.02f ",h_std_dev_arr[i]);
//}

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
float mean = (float)h_max[0]/actual_n;
reduce_diff<<<1, N>>>(d_std_dev_arr,mean);
hipMemcpy(h_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

//printf("\n");
//for(int i =0;i<N;i++)
//printf("%0.02f ",h_std_dev_arr[i]);

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
square_reduce<<<1,N>>>(d_std_dev_arr);
hipMemcpy(h_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

//printf("\n");
//for(int i =0;i<N;i++)
//printf("%0.02f ",h_std_dev_arr[i]);

float *h_sum_std_dev_arr;
h_sum_std_dev_arr = (float*)malloc(sizeof(float)*N);

hipMemcpy(d_std_dev_arr, h_std_dev_arr, sizeof(float)*N, hipMemcpyHostToDevice);
reduce_max<<<1, N/2>>>(d_std_dev_arr);
hipMemcpy(h_sum_std_dev_arr, d_std_dev_arr, sizeof(float)*N, hipMemcpyDeviceToHost);

//printf("\n");
//printf("%0.02f\n",h_sum_std_dev_arr[0]);

printf("Variance is %0.02f\n",h_sum_std_dev_arr[0]/actual_n);
printf("Std Deviance is %0.02f\n", sqrt(h_sum_std_dev_arr[0]/actual_n));
printf("Time taken %f\n", omp_get_wtime() - start);
*/
return 0;  
}
