#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 2048

//Note: N should always be in powers of 2 (like 2, 4, 8, 16, 32, ...) -Mohit Agrawal

__global__ void FindMin(int* input)
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			if(input[fst] >= input[snd])
			{
				input[fst] = input[snd];
			}
			else
			{
				input[fst] = input[fst];
			}
		}
		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}
int main()
{
	time_t t;
	srand((unsigned) time(&t));
	
	int *h;
	h = (int*)malloc(N*sizeof(int));
	
	for(int i=0; i<N; i++)
	{
		h[i] = rand()%N;
	}
	for(int i=0; i<N; i++)
	{
		printf("%d ", h[i]);
	}
	printf("\n");

	int* d;
	hipMalloc(&d, N*sizeof(int));
	
	hipMemcpy(d, h, N*sizeof(int), hipMemcpyHostToDevice);

	FindMin <<<1, N/2 >>>(d);
	
	hipDeviceSynchronize();

	int *result;
	result = (int*)malloc(sizeof(int));
	
	hipMemcpy(result, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("Min is: %d \n", result[0]);

	hipFree(d);
	free(h);

	return 0;
}
