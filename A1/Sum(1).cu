#include "hip/hip_runtime.h"
#include <stdio.h>
#define N 2048

//Note: N should always be in powers of 2 (like 2, 4, 8, 16, 32, ...) -Mohit Agrawal

__global__ void FindSum(int input[])
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			input[fst] += input[snd];
		}
		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}
int main()
{
	time_t t;
	srand((unsigned) time(&t));
	
	int *h;
	h = (int*)malloc(N*sizeof(int));
	
	for(int i=0; i<N; i++)
	{
		h[i] = rand()%N;
	}
	for(int i=0; i<N; i++)
	{
		printf("%d ", h[i]);
	}
	printf("\n");

	int* d;
	hipMalloc(&d, N*sizeof(int));
	
	hipMemcpy(d, h, N*sizeof(int), hipMemcpyHostToDevice);

	FindSum <<<1, N/2 >>>(d);
	
	hipDeviceSynchronize();

	int *result;
	result = (int*)malloc(sizeof(int));
	
	hipMemcpy(result, d, sizeof(int), hipMemcpyDeviceToHost);

	printf("Sum: %d \n", result[0]);

	hipFree(d);
	free(h);
	
	return 0;
}
