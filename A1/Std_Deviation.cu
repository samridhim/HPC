#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#define N 1024

//Note: N should always be in powers of 2 (like 2, 4, 8, 16, 32, ...) -Mohit Agrawal

__global__ void FindSum(float input[])
{
	int tid = threadIdx.x;
	int step_size = 1;
	int number_of_threads = blockDim.x;

	while (number_of_threads > 0)
	{
		if (tid < number_of_threads)
		{
			int fst = tid * step_size * 2;
			int snd = fst + step_size;
			input[fst] += input[snd];
		}
		step_size <<= 1; 
		number_of_threads >>= 1;
	}
}
__global__ void FindDiff(float input[], float mean)
{
	int tid = threadIdx.x;
	
	if (tid < N)
	{
		input[tid] = input[tid] - mean;
	}
}
int main()
{
	//Initialization
	time_t t;
	srand((unsigned) time(&t));
	
	float *h;
	h = (float*)malloc(N*sizeof(float));
	
	for(int i=0; i<N; i++)
	{
		h[i] = ((float)rand() / (float)RAND_MAX) * N;
	}
	for(int i=0; i<N; i++)
	{
		printf("%f ", h[i]);
	}
	printf("\n");

	//Finding sum
	float* d;
	hipMalloc(&d, N*sizeof(float));
	hipMemcpy(d, h, N*sizeof(float), hipMemcpyHostToDevice);

	FindSum <<<1, N/2 >>>(d);
	hipDeviceSynchronize();

	float *result;
	result = (float*)malloc(sizeof(float));
	hipMemcpy(result, d, sizeof(float), hipMemcpyDeviceToHost);
	printf("Sum is: %f \n", result[0]);
	
	//Mean calculation
	float avg = result[0]/N;
	printf("Avg is: %f \n", avg);

	//Subtracting mean from each element
	float *g;
	hipMalloc(&g, N*sizeof(float));
	hipMemcpy(g, h, N*sizeof(float), hipMemcpyHostToDevice);
	
	FindDiff <<<1, N >>>(g, avg);
	hipDeviceSynchronize();
	
	float *solution;
	solution = (float*)malloc(N*sizeof(float));
	hipMemcpy(solution, g, N*sizeof(float), hipMemcpyDeviceToHost);
	
	printf("Difference: ");
	for(int i=0; i<N; i++)
	{
		printf("%f ", solution[i]);
	}
	printf("\n");
	
	for(int i=0; i<N; i++)
	{
		solution[i] = fabsf(solution[i] * solution[i]);
	}
	
	printf("Squares: ");
	for(int i=0; i<N; i++)
	{
		printf("%f ", solution[i]);
	}
	printf("\n");
	
	float *solute;
	hipMalloc(&solute, N*sizeof(float));
	hipMemcpy(solute, solution, N*sizeof(float), hipMemcpyHostToDevice);
	
	//Adding the squares of differences
	FindSum <<<1, N/2 >>>(solute);
	hipDeviceSynchronize();

	float *std_dev;
	std_dev = (float*)malloc(sizeof(float));
	hipMemcpy(std_dev, solute, sizeof(float), hipMemcpyDeviceToHost);

	printf("Sum of Squares: ");
	printf("%f \n", std_dev[0]);
	
	//Taking arithmetic mean of the sqaures of differences
	float sol = std_dev[0]/N;
	
	float answer = sqrt(sol);
	
	printf("Standard Deviation is: %f \n", answer);
	
	hipFree(d);
	free(h);

	return 0;
}
