
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
__global__ void vectadd(float *a, float *b, float *c, int n, int m){
int tid = threadIdx.x + blockIdx.x * blockDim.x;
int sum =0;
if(tid<m){
for(int i =0;i<n;i++){
sum+= a[i]*b[i*m+tid];
}
c[tid] = sum;
}
}

int main(){
float *a, *b, *c;
float *d_a, *d_b, *d_c;

int n = 3;
int m = 4;

a = (float *) malloc(sizeof(float)*n);
b = (float *)malloc(sizeof(float)*n*m);
c = (float *)malloc(sizeof(float)*m);

hipMalloc((void **)&d_a, sizeof(float)*n);
hipMalloc((void **)&d_b, sizeof(float)*n*m);
hipMalloc((void **)&d_c, sizeof(float)*m);

for(int i =0;i<n;i++){
a[i] = rand()%10;
printf("%f ", a[i]);
}
printf("\n");
for(int i =0;i<n;i++){
for(int j =0;j<m;j++){
b[i*m +j] = rand()%9;
printf("%f ",b[i*n +j]);
}
printf("\n");
}
printf("\n");

hipMemcpy(d_a, a, sizeof(float)*n,hipMemcpyHostToDevice);
hipMemcpy(d_b, b, sizeof(float)*n*m, hipMemcpyHostToDevice);

vectadd<<<ceil(float(4/3)), 4>>>(d_a, d_b, d_c, n, m);
hipMemcpy(c, d_c, sizeof(float)*m, hipMemcpyDeviceToHost);

for(int i =0;i<m;i++){
printf("%f", c[i]);}

return 0;
}
