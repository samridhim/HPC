
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 3
#define NUM_THREADS_PER_BLOCK 256
__global__ void mm_kernel(float *d_a, float *d_b, float *d_c, int n){

int op_index = blockDim.x * blockIdx.x + threadIdx.x;
  int row = op_index / n, col = op_index % n;
  if(op_index<n*n)
    {
      float temp=0;
      for(int i=0; i<n; i++)
	temp += d_a[row*n+i] * d_b[col + i*n];
      d_c[op_index] = temp;
    }
}

int main(){
float *a, *b, *c;
float *d_a, *d_b, *d_c;
int n = N*N;

a = (float *)malloc(n);
b = (float *)malloc(n);
c = (float *)malloc(n);

hipMalloc((void **)&d_a,sizeof(float)*n);
hipMalloc((void **)&d_b,sizeof(float)*n);
hipMalloc((void **)&d_c,sizeof(float)*n);

srand(time(NULL));
for(int i =0;i<N;i++){
 for(int j =0;j<N;j++){
	a[i*N +j] = rand()%10; 	
	printf("%f ", a[i*N+j]);
}
printf("\n");
}
printf("\n\n");

srand(time(NULL));
for(int i =0;i<N;i++){
 for(int j =0;j<N;j++){
	b[i*N +j] = rand()%9; 
	printf("%f ", b[i*N+j]);	
}
printf("\n");
}

printf("\n\n");
hipMemcpy(d_a, a, sizeof(float)*n, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, sizeof(float)*n, hipMemcpyHostToDevice);

mm_kernel<<<ceil((float)N*N/NUM_THREADS_PER_BLOCK), NUM_THREADS_PER_BLOCK>>>(d_a, d_b, d_c, N);
hipMemcpy(c, d_c, sizeof(float)*n, hipMemcpyDeviceToHost);
for(int i =0;i<N;i++){
 for(int j =0;j<N;j++){ 
	printf("%f ", c[i*N+j]);	
}
printf("\n");
}
hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
