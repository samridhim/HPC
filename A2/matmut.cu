
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<time.h>
#define N 2
#define M 3
#define P 2
#define BLOCK_SIZE 16
__global__ void mm_kernel(float *d_a, float *d_b, float *d_c, int n, int m, int p){
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < p && row < n) 
    {
        for(int i = 0; i < m; i++) 
        {
            sum += d_a[row * m + i] * d_b[i * p + col];
        }
        d_c[row * p + col] = sum;
}
}

int main(){
float *a, *b, *c;
float *d_a, *d_b, *d_c;
int n = N*M;
int t = M*P;
int u = N*P;

a = (float *)malloc(n);
b = (float *)malloc(t);
c = (float *)malloc(u);

hipMalloc((void **)&d_a,sizeof(float)*n);
hipMalloc((void **)&d_b,sizeof(float)*t);
hipMalloc((void **)&d_c,sizeof(float)*u);

srand(time(NULL));
for(int i =0;i<N;i++){
 for(int j =0;j<M;j++){
	a[i*M +j] = rand()%10; 	
	printf("%f ", a[i*M+j]);
}
printf("\n");
}
printf("\n\n");

srand(time(NULL));
for(int i =0;i<M;i++){
 for(int j =0;j<P;j++){
	b[i*P +j] = rand()%9; 
	printf("%f ", b[i*P+j]);	
}
printf("\n");
}

printf("\n\n");
hipMemcpy(d_a, a, sizeof(float)*n, hipMemcpyHostToDevice);
hipMemcpy(d_b, b, sizeof(float)*t, hipMemcpyHostToDevice);
mm_kernel<<<dim3(BLOCK_SIZE,BLOCK_SIZE,1), dim3(N,P,1)>>>(d_a, d_b, d_c, N,M,P);
hipMemcpy(c, d_c, sizeof(float)*u, hipMemcpyDeviceToHost);
for(int i =0;i<N;i++){
 for(int j =0;j<P;j++){ 
	printf("%f ", c[i*P+j]);	
}
printf("\n");
}
hipFree(d_a); hipFree(d_b); hipFree(d_c);
}
