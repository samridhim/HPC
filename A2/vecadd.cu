
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<time.h>
#include"omp.h"
#define N 1024
#define threads_per_block 1024
__global__ void add(float*a,float*b,float*c)
{
 int tid = threadIdx.x + blockIdx.x*blockDim.x;
 c[tid] = a[tid] + b[tid];
}
int main()
{
	int num_blocks;
	float start = clock();
	float *a, *b, *c;
	float *d_a, *d_b, *d_c;
	int n = sizeof(float)*N;
	hipMalloc((void **)&d_a, n);
	hipMalloc((void **)&d_b, n);
	hipMalloc((void **)&d_c, n);
	a = (float *)malloc(n);
	b = (float *)malloc(n);
	c = (float *)malloc(n);
	
	printf("A and B are \n");
	srand(time(NULL));	
	for(int i =0;i<N;i++)
	{
	a[i] = 1;
	printf("%0.03f ", a[i]);	
	}	
	printf("\n\n");
	srand(time(NULL));	
	for(int i =0;i<N;i++)
	{
	b[i] = 1;
	printf("%0.03f ", b[i]);
	}	
	printf("\n\n");
	hipMemcpy(d_a, a,n, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b,n, hipMemcpyHostToDevice);
	if(N/threads_per_block ==0)
	num_blocks = 1;
	else
	num_blocks = N/threads_per_block;
	add<<<num_blocks,threads_per_block>>>(d_a, d_b, d_c);
	hipMemcpy(c, d_c,n, hipMemcpyDeviceToHost);
	for(int i =0;i<N;i++)
	printf("%0.02f ", c[i]);	
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	printf("\n\n Time taken %f\n", (double)(clock()-start)/CLOCKS_PER_SEC);	

 	 return 0;
}
